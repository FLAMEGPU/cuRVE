#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include "hip/hip_runtime.h"
#include "curve/curve.cuh"




__global__ void vectorAdd(int numElements)
{
    unsigned int idx;
	float a, b, c;

    idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    a = getFloatVariable("a", idx);
    b = getFloatVariable("b", idx);
    //printf("B is %f \n", b);

    c = a + b;

    setFloatVariable("c", c, idx);

    curveReportLastDeviceError();
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int numElements = 256;

    curveInit(numElements);
    curveSetNamespace("name1");
    curveRegisterVariable("a");
    curveRegisterVariable("b");
    curveRegisterVariable("c");
    curveReportErrors();

    printf("[Vector addition of %d elements]\n", numElements);


    for (int i = 0; i < numElements; ++i)
	{
    	float a = rand()/(float)RAND_MAX;
		float b = (float) i;
    	curveSetFloat("a", a, i);
		curveSetFloat("b", b, i);
		curveReportErrors();
	}

    //timing start
    hipEvent_t start,stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(numElements);
    err = hipGetLastError();

    //timing end
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);

    printf("Kernel Time was %f ms\n", elapsedTime);

    curveReportErrors();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
